#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "hip/hip_runtime_api.h"
#include "wtime.h"
#include "params.h"

template <typename T>
T div_ceil(T a, T b) {
    return (a + b - 1) / b;
}

__global__ void setup_prng(hiprandState * prng_states) {
  int id = threadIdx.x + blockIdx.x * blockDim.x; 

  /* Copy state to local memory for efficiency */ 
  hiprandState localState = prng_states[id]; 

  /* Each thread gets same seed, a different sequence
     number, no offset */
  hiprand_init(1234, id, 0, &prng_states[id]);
}

__global__ void photon(float * heat, hiprandState * prng_states) {
    unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;
    hiprandState prng_state = prng_states[id];

    int thread_from = WORK * (blockDim.x * blockIdx.x + threadIdx.x);
    int thread_to = min(WORK * (blockDim.x * blockIdx.x + threadIdx.x + 1),
            PHOTONS);

    const float albedo = MU_S / (MU_S + MU_A);
    const float shells_per_mfp = 1e4 / MICRONS_PER_SHELL / (MU_A + MU_S);

    /* launch */
    float x = 0.0f;
    float y = 0.0f;
    float z = 0.0f;
    float u = 0.0f;
    float v = 0.0f;
    float w = 1.0f;
    float weight = 1.0f;

    float _heat[SHELLS] = {0}; // local heat

    for (int id = thread_from; id < thread_to; ++id)
    {
        for (;;) {
            /* move */
            float t = -logf(hiprand_uniform(prng_state));
            x += t * u;
            y += t * v;
            z += t * w;

            /* absorb */
            unsigned int shell = sqrtf(x * x + y * y + z * z) * shells_per_mfp;
            if (shell > SHELLS - 1) {
                shell = SHELLS - 1;
            }

            _heat[shell] += (1.0f - albedo) * weight;
            weight *= albedo;

            /* roulette */
            if (weight < 0.001f) { 
                if (hiprand_uniform(prng_state) > 0.1f)
                    break;
                weight /= 0.1f;
            }

            /* New direction, rejection method */
            float xi1, xi2;
            do {
                xi1 = 2.0f * hiprand_uniform(prng_state) - 1.0f;
                xi2 = 2.0f * hiprand_uniform(prng_state) - 1.0f;
                t = xi1 * xi1 + xi2 * xi2;
            } while (1.0f < t); 
            u = 2.0f * t - 1.0f;
            v = xi1 * sqrtf((1.0f - u * u) / t);
            w = xi2 * sqrtf((1.0f - u * u) / t);
        }
    }

    for (int i = 0; i < SHELLS; i++) {
        atomicAdd(&heat[i], _heat[i]);
    }
}

int main() {

    // histogram
    float * heat;
    checkCudaCall(hipMallocManaged(&heat, SHELLS * sizeof(float)));

    // gpu timers
    hipEvent_t gpu_start, gpu_finish; // timers
    checkCudaCall(hipEventCreate(&gpu_start));
    checkCudaCall(hipEventCreate(&gpu_finish));

    // kernel parameters
    const unsigned int threads_per_block = BLOCK_SIZE;
    const unsigned int block_count = div_ceil(PHOTONS, BLOCK_SIZE * WORK);
    const unsigned int total_threads = threads_per_block * block_count;

    // seed/PRNG setup
    hiprandState *prng_states;
    checkCudaCall((hipMalloc((void **)&devStates,
                    totalThreads * sizeof(hiprandState))); 

    setup_prng<<<block_count, threads_per_block>>>(prng_states);
    checkCudaCall(hipGetLastError());

    // launch kernel
    checkCudaCall(hipEventRecord(gpu_start));
    photon<<<block_count, threads_per_block>>>(heat, prng_states);
    checkCudaCall(hipGetLastError());
    checkCudaCall(hipEventRecord(gpu_finish));
    checkCudaCall(hipDeviceSynchronize());

    // elapsed time
    float gpu_elapsed;
    checkCudaCall(hipEventElapsedTime(&gpu_elapsed, gpu_start, gpu_finish));
    checkCudaCall(hipEventDestroy(gpu_start));
    checkCudaCall(hipEventDestroy(gpu_finish));

    // heat2 calc
    float heat2[SHELL] = {0};
    for (int i = 0; i < SHELLS; i++) {
        heat2[i] = heat[i] * heat[i];
    }

    // output
#if VERBOSE
    printf("# Scattering = %8.3f/cm\n", MU_S);
    printf("# Absorption = %8.3f/cm\n", MU_A);
    printf("# Photons    = %8d\n#\n", PHOTONS);

    printf("# %lf K photons per second\n", 1e-3 * PHOTONS / elapsed);
    printf("# %lf seconds\n", elapsed);
    printf("# Radius\tHeat\n");
    printf("# [microns]\t[W/cm^3]\tError\n");

    float t = 4.0f * M_PI * powf(MICRONS_PER_SHELL, 3.0f) * PHOTONS / 1e12;
    for (unsigned int i = 0; i < SHELLS - 1; ++i) {
        printf("%6.0f\t%12.5f\t%12.5f\n", i * (float)MICRONS_PER_SHELL,
               heat[i] / t / (i * i + i + 1.0 / 3.0),
               sqrt(heat2[i] - heat[i] * heat[i] / PHOTONS) / t /
               (i * i + i + 1.0f / 3.0f));
    }
    printf("# extra\t%12.5f\n", heat[SHELLS - 1] / PHOTONS);
#else // copiar lo que está abajo arriba cuando no haya mas cosas que agregar
    printf("block size: %u\n", BLOCK_SIZE);
    printf("grid size: %u\n", div_ceil(PHOTONS, BLOCK_SIZE * WORK));
    printf("total threads: %u\n", BLOCK_SIZE * 
            div_ceil(PHOTONS, BLOCK_SIZE * WORK);
    printf("device allocated : % lu\n", sizeof(hiprandState) * total_threads);
    printf("%photons: lf\n", PHOTONS);
    printf("%ph/s: lf\n", 1e-3 * PHOTONS / gpu_elapsed);
    printf("gpu: %f ms\n", gpu_elapsed);
    // printf("total: %f ms\n", elapsed);
#endif

    // cleanup
    checkCudaCall(hipFree(heat));
    checkCudaCall(hipFree(prng_states));

    return 0;
}
