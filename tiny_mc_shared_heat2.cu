#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdint>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "hip/hip_runtime_api.h"
#include "wtime.h"
#include "params.h"

template <typename T>
T div_ceil(T a, T b) {
    return (a + b - 1) / b;
}

__global__ void setup_prng(hiprandState * prng_states) {
    int id = threadIdx.x + blockIdx.x * blockDim.x; 
    hiprand_init(1234, id, 0, &prng_states[id]);
}

__global__ void photon(float * heat, float * heat2, hiprandState * prng_states) {
    unsigned int id = blockDim.x * blockIdx.x + threadIdx.x;

    hiprandState prng_state = prng_states[id];

    int thread_from = WORK * (blockDim.x * blockIdx.x + threadIdx.x);
    int thread_to = min(WORK * (blockDim.x * blockIdx.x + threadIdx.x + 1),
            PHOTONS);

    const float albedo = MU_S / (MU_S + MU_A);
    const float shells_per_mfp = 1e4 / MICRONS_PER_SHELL / (MU_A + MU_S);

    __shared__ float _heat[SHELLS], _heat2[SHELLS];

    // fist thread inits shared heat (per block)
    if (threadIdx.x == 0) {
        for (int i = 0; i < SHELLS; i++) {
            _heat[i] = 0;
            _heat2[i] = 0;
        }
    }

    __syncthreads(); // wait for initialization

    for (int id = thread_from; id < thread_to; ++id)
    {
        /* launch */
        float x = 0.0f;
        float y = 0.0f;
        float z = 0.0f;
        float u = 0.0f;
        float v = 0.0f;
        float w = 1.0f;
        float weight = 1.0f;
        for (;;) {
            /* move */
            float t = -logf(hiprand_uniform(&prng_state));
            x += t * u;
            y += t * v;
            z += t * w;

            /* absorb */
            unsigned int shell = sqrtf(x * x + y * y + z * z) * shells_per_mfp;
            if (shell > SHELLS - 1) {
                shell = SHELLS - 1;
            }

            atomicAdd(&_heat[shell], (1.0f - albedo) * weight);
            atomicAdd(&_heat2[shell], (1.0f - albedo) * (1 - albedo) * weight * weight);
            weight *= albedo;

            /* roulette */
            if (weight < 0.001f) { 
                if (hiprand_uniform(&prng_state) > 0.1f)
                    break;
                weight /= 0.1f;
            }

            /* New direction, rejection method */
            float xi1, xi2;
            do {
                xi1 = 2.0f * hiprand_uniform(&prng_state) - 1.0f;
                xi2 = 2.0f * hiprand_uniform(&prng_state) - 1.0f;
                t = xi1 * xi1 + xi2 * xi2;
            } while (1.0f < t); 
            u = 2.0f * t - 1.0f;
            v = xi1 * sqrtf((1.0f - u * u) / t);
            w = xi2 * sqrtf((1.0f - u * u) / t);
        }
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        for (int i = 0; i < SHELLS; i++) {
            atomicAdd(&heat[i], _heat[i]);
            atomicAdd(&heat2[i], _heat2[i]);
        }
    }
}

int main() {

    // histograms
    float *heat, *heat2;
    checkCudaCall(hipMallocManaged(&heat, SHELLS * sizeof(float)));
    checkCudaCall(hipMallocManaged(&heat2, SHELLS * sizeof(float)));
    for (int i = 0; i < SHELLS; i++) {
        heat[i] = 0;
        heat2[i] = 0;
    }

    // kernel parameters
    const unsigned int block_count = div_ceil(PHOTONS, BLOCK_SIZE * WORK);
    const unsigned int total_threads = BLOCK_SIZE * block_count;

    // PRNG setup
    hiprandState *prng_states;
    checkCudaCall(hipMalloc((void **)&prng_states,
                total_threads * sizeof(hiprandState))); 

    setup_prng<<<block_count, BLOCK_SIZE>>>(prng_states);
    checkCudaCall(hipGetLastError());

    // gpu timers
    hipEvent_t gpu_start, gpu_finish;
    checkCudaCall(hipEventCreate(&gpu_start));
    checkCudaCall(hipEventCreate(&gpu_finish));

    // cpu timers
    double cpu_start = wtime();

    // launch kernel
    checkCudaCall(hipEventRecord(gpu_start));
    photon<<<block_count, BLOCK_SIZE>>>(heat, heat2, prng_states);
    checkCudaCall(hipGetLastError());
    checkCudaCall(hipEventRecord(gpu_finish));
    checkCudaCall(hipDeviceSynchronize());

    // elapsed gpu time
    float gpu_elapsed;
    checkCudaCall(hipEventElapsedTime(&gpu_elapsed, gpu_start, gpu_finish));

    // elapsed cpu time (total time)
    double cpu_elapsed = wtime() - cpu_start;

    // output
#if VERBOSE
    printf("# Scattering          = %8.3f/cm\n", MU_S);
    printf("# Absorption          = %8.3f/cm\n", MU_A);
    printf("# Photons             = %8d\n", PHOTONS);
    printf("# Elapsed (Total)     = %lf ms\n", 1e3 * cpu_elapsed);
    printf("# Elapsed (GPU)       = %f ms\n", gpu_elapsed);
    printf("# Photons per second (Total) = %lf K\n", 1e-3 * PHOTONS / cpu_elapsed);
    printf("# Photons per second (GPU)   = %lf K\n\n", PHOTONS / gpu_elapsed);

    printf("# Radius\tHeat\n");
    printf("# [microns]\t[W/cm^3]\tError\n");
    float t = 4.0f * M_PI * powf(MICRONS_PER_SHELL, 3.0f) * PHOTONS / 1e12;
    for (unsigned int i = 0; i < SHELLS - 1; ++i) {
        printf("%6.0f\t%12.5f\t%12.5f\n", i * (float)MICRONS_PER_SHELL,
                heat[i] / t / (i * i + i + 1.0 / 3.0),
                sqrt(heat2[i] - heat[i] * heat[i] / PHOTONS) / t /
                (i * i + i + 1.0f / 3.0f));
    }

    printf("# extra\t%12.5f\n\n", heat[SHELLS - 1] / PHOTONS);
    printf("# GPU stats:\n");
    printf("Block size: %u\n", BLOCK_SIZE);
    printf("Grid size: %u\n", block_count);
    printf("Total threads: %u\n", total_threads);
    printf("Device allocated: %f GB\n", float(sizeof(hiprandState) * total_threads) / (1024 * 1024 * 1024));
    printf("Work: %d\n", WORK);
#else
    printf("# Photons per second (Total) = %lf K\n", 1e-3 * PHOTONS / cpu_elapsed);
#endif

    // cleanup
    checkCudaCall(hipEventDestroy(gpu_start));
    checkCudaCall(hipEventDestroy(gpu_finish));
    checkCudaCall(hipFree(heat));
    checkCudaCall(hipFree(prng_states));

    return 0;
}
